/**
* Author: 	Vishnu
* File: 	main.cu
* Purpose: 	Brings the entire project together. Inits a trie, 
* 			adds patterns to it, reads the input file
* 			detect patterns given in patterns.h (if any)
* 			and prints them stdout
**/

#include <stdio.h> 
#include <stdlib.h>
#include <string.h>

#include "../include/trie.h"
#include "../include/ac_str_match.h"

#include "../include/patterns.h"

#define MAX_INPUT_SZ 500

const char *input_file_path = "../files/input_stream.txt";

const char *pattern_list[] = {
	// 1.
	"bash -i >& /dev/tcp/",
	"subprocess.call",
	"sprintf",
	"exec",
	"nc",

	// 2.
	"/bin",
	"/sh",

	// 3.
	".pl",
	".php",
	".py",
	".sh"
};

const size_t NUMBER_OF_PATTERNS = sizeof(pattern_list) / sizeof(char *);

/**
* This function adds patterns to the initialized trie
**/
struct trie_node *add_patterns(struct trie_node *root) {
	for (int i=0; i<NUMBER_OF_PATTERNS; i++) {
		insert_pattern(root, pattern_list[i]);
	}
	return root;
}

void initializePatterns(char *patterns, int *indices) {
	int idx_cnt = 0, pat_cnt = 0;
	indices[idx_cnt++] = 0;
	for(int i=0;i<NUMBER_OF_PATTERNS;i++) {
		indices[idx_cnt] = indices[idx_cnt++-1] + strlen(pattern_list[i]);
		for(int j=0;j<strlen(pattern_list[i]);j++) {
			patterns[pat_cnt++] = pattern_list[i][j];
		}
	}
	patterns[pat_cnt] = '\0';
}

int main(int argc, char *argv[]) {

	struct trie_node *root = get_node(); 									// initialize a trie
	add_patterns(root); 													// add patterns from the header file

	char *input_buffer = (char *)malloc(sizeof(char) * MAX_INPUT_SZ);		// initialize buffer
	FILE *fhandle = fopen(input_file_path, "r");							// open file as read-only
	size_t ret = fread(input_buffer, sizeof(char), MAX_INPUT_SZ, fhandle); 	// read file into the buffer

	int* indices = (int*)malloc(sizeof(int) * (NUMBER_OF_PATTERNS+1));
	int* matches = (int*)calloc(NUMBER_OF_PATTERNS, sizeof(int));
	char *patterns = (char*)malloc(sizeof(char) * 1024);

	initializePatterns(patterns, indices);

	int *d_indices, *d_matches;
	char *d_patterns, *d_input;

	hipMalloc((void**)&d_indices, sizeof(int) * (NUMBER_OF_PATTERNS+1));
	hipMalloc((void**)&d_patterns, sizeof(char) * strlen(patterns));
	hipMalloc((void**)&d_input, sizeof(char) * ret);
	hipMalloc((void**)&d_matches, sizeof(int) * NUMBER_OF_PATTERNS);

	hipMemcpy(d_indices, indices, sizeof(int) * (NUMBER_OF_PATTERNS+1), hipMemcpyHostToDevice);
	hipMemcpy(d_matches, matches, sizeof(int) * NUMBER_OF_PATTERNS, hipMemcpyHostToDevice);
	hipMemcpy(d_patterns, patterns, sizeof(char) * strlen(patterns), hipMemcpyHostToDevice);
	hipMemcpy(d_input, input_buffer, sizeof(char) * ret, hipMemcpyHostToDevice);

	int len = strlen(input_buffer);
	dim3 grid_size(ret,1,1);
	dim3 block_size(NUMBER_OF_PATTERNS, 1, 1);
	
	detect_patterns<<<grid_size, block_size>>>(d_input, d_patterns, d_indices, d_matches, len);

	hipDeviceSynchronize();

	hipMemcpy(matches, d_matches, sizeof(int) * NUMBER_OF_PATTERNS, hipMemcpyDeviceToHost);

	char buffer[1024];
	int cnt = 0;
	for(int i=0;i<NUMBER_OF_PATTERNS;i++) {
		if(matches[i] == 1) {
			cnt = 0;
			for(int j=indices[i];j<indices[i+1];j++) {
				buffer[cnt++] = patterns[j];
			}
			buffer[cnt] = '\0';
			printf("Pattern %s has been found!\n", buffer);
		}
	}
	
	hipFree(d_input);
	hipFree(d_patterns);
	hipFree(d_indices);
	hipFree(d_matches);

	free(input_buffer);
	free(patterns);
	free(indices);
	free(matches);

	return 0;
}
