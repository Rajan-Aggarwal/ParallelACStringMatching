#include "hip/hip_runtime.h"
/**
* Author: 	Vishnu
* File: 	ac_str_match.cu
* Purpose: 	src file for the implementation of the accelerated 
*			AC string matching algorithm
**/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../include/trie.h"
#include "../include/ac_str_match.h"

__global__ void detect_patterns(char *input, char *patterns, int *indices, int *matches, int len) {
	int start = blockIdx.x;
	int idx = threadIdx.x;

	int beg = indices[idx], end = indices[idx+1];

	if((len-start+1) >= (end-beg+1)) {
		int index = start;
		for(int i=beg;i<end;i++) {
			if(patterns[i] != input[index++]) {
				return;
			}
		}
		matches[idx] = 1;
	}
}
